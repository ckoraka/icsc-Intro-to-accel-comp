#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

//Set 2-D Matrix size: SIZE*SIZE
const int DSIZE = 256;
const int a = 1;
const int b = 2;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)

// Check if matrix multiplication was correct
int check_result(const int *C){

	for (int i = 0; i < DSIZE*DSIZE; i++) {
		if (C[i] != a*b*DSIZE) {
			printf("Error : Index %d was %d instead of %d\n", i, C[i], a*b*DSIZE);
			return -1;
		}
	}
	printf("Matrix multiplication was correct!\n");
	return 0;
}

// Function that runs on the CPU
void matrix_mult_cpu(const int *A, const int *B, int *C, int N) {

	for(int i=0; i<N; ++i)
	{
		for(int j=0; j<N; ++j)
		{
			int sum = 0; 
			for(int k=0; k<N; ++k)
				sum += A[k+N*i] * B[j+k*N];
			C[j+N*i] = sum;
		}
	}
}

// CUDA kernel that runs on the GPU
__global__ void matrix_mult_gpu(const int *A, const int *B, int *C, int N) {

	// Express the matrix index in 2-D as a function of the threadIdx and the blockIdx
    int idx = FIXME;
    int idy = FIXME;

	if(idx<N && idy<N){
		int sum = 0; 
		for(int k=0; k<N; k++)
			FIXME : Write the dot product using idx,idy,k and N
		C[FIXME] = sum; 
	}
}

int main() {

	// Variables used to measure time
	clock_t t0, t1, t2, t3;
	double t_cpu = 0.0;
	double t_gpu = 0.0;

	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE*DSIZE];
	h_B = new int[DSIZE*DSIZE];
	h_C = new int[DSIZE*DSIZE];
	for (int i = 0; i < DSIZE*DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
		h_C[i] = 0;
	}

	// Measure time on CPU : Start timing
	t0 = clock();

	// Call the CPU function
	FIXME
	// Calculate & print CPU time
	t1 = clock();
	t_cpu = ((double)(t1-t0))/CLOCKS_PER_SEC;
	printf ("CPU took %f seconds\n", t_cpu);

	// Check if matrix multiplication on CPU was correct
	check_result(h_C);

	// Initialize host pointer that holds result
	for (int i = 0; i < DSIZE*DSIZE; i++)
		h_C[i] = 0;

	// Measure time on GPU : Start timing
	t2 = clock();

	// Allocate device memory 
	hipMalloc(&d_A, DSIZE*DSIZE*sizeof(int));
	FIXME : Do the same for the additional device pointers

	// Check memory allocation for errors
	cudaCheckErrors();

	// Copy the matrices on GPU
	hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(int), hipMemcpyHostToDevice);
	FIXME : Copy h_B from host to device
	// Check memory copy for errors
	cudaCheckErrors();

	// Define the number of threads per block
	int blockDim = 32;
	// dim3: Native CUDA type used to specify dimensions (up to 3 arguments)
	dim3 block(blockDim, blockDim);
	// Define the number of blocks in the grid
	dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);
	// Launch kernel
	matrix_mult_gpu<<<FIXME,FIXME>>>(FIXME);

	// Check kernel launch for errors
	FIXME

	// Copy results back to host
	FIXME

	// Measure time on GPU
	FIXME

	// Check if matrix multiplication on GPU was correct
	FIXME

	// Free the allocated memory 
	FIXME
	
	return 0;

}